#include "hip/hip_runtime.h"
#include "FilterByConfidenceCuda.h"
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>

// CUDA kernel
__global__ void filterByConfidenceKernel(int classesNumber, int vectorSize,
                                         float *transOutputVector,
                                         int *indexes,
                                         float *class_ind,
                                         float *conf_vector,
                                         float confThreshold, 
                                         int *outCount) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global index
    if (i < vectorSize) {
        float max_element = 0.0;
        float ind = -1.0; // Initialize to -1 to signify no class

        for (int j = 4; j < (classesNumber + 4); ++j) {
            float current_value = transOutputVector[i * 84 + j];
            if (max_element < current_value) { // Flatten the 2D array
                max_element = current_value;
                ind = static_cast<float>(j); // Save the index of the max class
            }
        }
        if (max_element > confThreshold) {
            int count = atomicAdd(outCount, 1); // Use atomic counter for output index
            indexes[count] = i; // Store index
            class_ind[count] = ind - 4.0f; // Store class index
            conf_vector[count] = max_element; // Store confidence
        }
    }
}

// Host function to initialize and call the CUDA kernel
void FilterByConfidenceCuda(int classesNumber, int vectorSize,
                             float transOutputVector[1][8400][84],
                             std::vector<int> &indexes,
                             std::vector<float> &class_ind,
                             std::vector<float> &conf_vector,
                             float confThreshold) {

    // Define sizes
    size_t transOutputVectorSize = vectorSize * 84 * sizeof(float);
    
    // Device pointers
    int *d_indexes;
    float *d_class_ind;
    float *d_conf_vector;
    float *d_transOutputVector;
    int *d_outCount;
    
    // Allocate device memory
    hipMalloc(&d_transOutputVector, transOutputVectorSize);
    hipMalloc(&d_indexes, vectorSize * sizeof(int));
    hipMalloc(&d_class_ind, vectorSize * sizeof(float));
    hipMalloc(&d_conf_vector, vectorSize * sizeof(float));
    hipMalloc(&d_outCount, sizeof(int));
    
    // Initialize output counter on device
    int zero = 0;
    hipMemcpy(d_outCount, &zero, sizeof(int), hipMemcpyHostToDevice);
    
    // Copy data to device
    hipMemcpy(d_transOutputVector, transOutputVector, transOutputVectorSize, hipMemcpyHostToDevice);
    
    // Launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (vectorSize + threadsPerBlock - 1) / threadsPerBlock;
    filterByConfidenceKernel<<<blocksPerGrid, threadsPerBlock>>>(classesNumber, vectorSize,
        d_transOutputVector, d_indexes, d_class_ind, d_conf_vector, confThreshold, d_outCount);
    
    // Copy results back to host
    int outCount;
    hipMemcpy(&outCount, d_outCount, sizeof(int), hipMemcpyDeviceToHost);
    
    indexes.resize(outCount);
    class_ind.resize(outCount);
    conf_vector.resize(outCount);
    
    hipMemcpy(indexes.data(), d_indexes, outCount * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(class_ind.data(), d_class_ind, outCount * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(conf_vector.data(), d_conf_vector, outCount * sizeof(float), hipMemcpyDeviceToHost);
    
    // Free device memory
    hipFree(d_transOutputVector);
    hipFree(d_indexes);
    hipFree(d_class_ind);
    hipFree(d_conf_vector);
    hipFree(d_outCount);
}